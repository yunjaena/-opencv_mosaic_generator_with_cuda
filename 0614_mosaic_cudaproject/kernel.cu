#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"

#include "C:\ProgramData\NVIDIA Corporation\CUDA Samples\v10.2\common\inc\hip/hip_runtime_api.h"

#include <hip/device_functions.h>

#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/core/cuda.hpp>
#include <omp.h>
#include <iostream>
#include "DS_definitions.h"
#include "DS_timer.h"

using namespace cv;
using namespace std;

__global__ void mosaic_cuda(unsigned char* in, unsigned char* out, int mon, int height, int width) {



    // cv::cuda::GpuMat image(*img);
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iy = blockDim.y * blockIdx.y + threadIdx.y;


    if (ix % mon != 0 || iy % mon != 0) return;

    if (ix >= height || iy >= width) {

        return;
    }
    int R, G, B;

    //R = in[idx * 3 + 0];
    //G = in[idx * 3 + 1];
    //B = in[idx * 3 + 2];


    int nCount;
    int monWidth = mon, monHeight = mon;



    nCount = 0;
    B = 0; G = 0; R = 0;

    if ((ix + mon) > height)
        monHeight = height % mon;
    else
        monHeight = mon;

    if ((iy + mon) > width)
        monWidth = width % mon;
    else
        monWidth = mon;

    for (int n = 0; n < monHeight; n++) {
        for (int m = 0; m < monWidth; m++) {
            int idx = (ix + n) * width + (iy + m);
            B += in[idx * 3 + 0];
            G += in[idx * 3 + 1];
            R += in[idx * 3 + 2];
            //nCount++;
        }
    }

    nCount = monHeight * monWidth;
    //���������

    B /= nCount;
    G /= nCount;
    R /= nCount;

    //B = 125;
    //G = 125;
    //R = 125;


    for (int n = 0; n < monHeight; n++) {
        for (int m = 0; m < monWidth; m++) {
            int idx = (ix + n) * width + (iy + m);

            out[idx * 3 + 0] = B;
            out[idx * 3 + 1] = G;
            out[idx * 3 + 2] = R;
        }
    }

}

__global__ void mosaic_cuda_sm(unsigned char* in, unsigned char* out, int mon, int height, int width) {



    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iy = blockDim.y * blockIdx.y + threadIdx.y;


    if (ix % mon != 0 || iy % mon != 0) return;

    if (ix >= height || iy >= width) {

        return;
    }
    __shared__ int R;
    __shared__ int G;
    __shared__ int B;

    int nCount;
    int monWidth = mon, monHeight = mon;



    nCount = 0;
    B = 0; G = 0; R = 0;
    __syncthreads();

    if ((ix + mon) > height)
        monHeight = height % mon;
    else
        monHeight = mon;

    if ((iy + mon) > width)
        monWidth = width % mon;
    else
        monWidth = mon;

    for (int n = 0; n < monHeight; n++) {
        for (int m = 0; m < monWidth; m++) {
            int idx = (ix + n) * width + (iy + m);
            B += in[idx * 3 + 0];
            G += in[idx * 3 + 1];
            R += in[idx * 3 + 2];
        }
    }

    nCount = monHeight * monWidth;

    B /= nCount;
    G /= nCount;
    R /= nCount;

    for (int n = 0; n < monHeight; n++) {
        for (int m = 0; m < monWidth; m++) {
            int idx = (ix + n) * width + (iy + m);

            out[idx * 3 + 0] = B;
            out[idx * 3 + 1] = G;
            out[idx * 3 + 2] = R;
        }
    }

}



void mosaic_serial(const Mat* image, Mat* dst, int mon)
{
    int i, j, n, m;

    int nCount;
    int monWidth = mon, monHeight = mon;

    int R, G, B;
    for (i = 0; i < image->size().height; i += mon) {
        for (j = 0; j < image->size().width; j += mon) {
            nCount = 0;
            B = 0; G = 0; R = 0;

            if ((i + mon) > image->size().height)
                monHeight = image->size().height % mon;
            else
                monHeight = mon;

            if ((j + mon) > image->size().width)
                monWidth = image->size().width % mon;
            else
                monWidth = mon;

            for (n = 0; n < monHeight; n++) {
                for (m = 0; m < monWidth; m++) {
                    B += image->at<Vec3b>(i + n, j + m)[0];
                    G += image->at<Vec3b>(i + n, j + m)[1];
                    R += image->at<Vec3b>(i + n, j + m)[2];
                    nCount++;
                }
            }

            //���������
            B /= nCount;
            G /= nCount;
            R /= nCount;

            for (n = 0; n < monHeight; n++) {
                for (m = 0; m < monWidth; m++) {
                    dst->at<Vec3b>(i + n, j + m)[0] = B;
                    dst->at<Vec3b>(i + n, j + m)[1] = G;
                    dst->at<Vec3b>(i + n, j + m)[2] = R;
                }
            }
        }
    }
}

void mosaic_openmp(const Mat* image, Mat* dst, int mon)
{
    const int NUM_THREADS = 6;
    for (int i = 0; i < image->size().height; i += mon) {
#pragma omp parallel for num_threads(NUM_THREADS)
        for (int j = 0; j < image->size().width; j += mon) {
            int B = 0, G = 0, R = 0;
            int monWidth, monHeight;

            if ((i + mon) > image->size().height)
                monHeight = image->size().height % mon;
            else
                monHeight = mon;

            if ((j + mon) > image->size().width)
                monWidth = image->size().width % mon;
            else
                monWidth = mon;

            for (int n = 0; n < monHeight; n++) {
                for (int m = 0; m < monWidth; m++) {
                    B += image->at<Vec3b>(i + n, j + m)[0];
                    G += image->at<Vec3b>(i + n, j + m)[1];
                    R += image->at<Vec3b>(i + n, j + m)[2];
                }
            }

            //���������
            int nCount = monHeight * monWidth;
            B /= nCount;
            G /= nCount;
            R /= nCount;

            for (int n = 0; n < monHeight; n++) {
                for (int m = 0; m < monWidth; m++) {
                    dst->at<Vec3b>(i + n, j + m)[0] = B;
                    dst->at<Vec3b>(i + n, j + m)[1] = G;
                    dst->at<Vec3b>(i + n, j + m)[2] = R;
                }
            }
        }
    }
}

#define mosaic 40

int main()
{


    string fileName;
    Mat original_image, result_serial, result_openmp;
    Mat* original_image_cuda;
    DS_timer timer(4);
    timer.setTimerName(0, (char*)"Serial");
    timer.setTimerName(1, (char*)"Openmp");
    timer.setTimerName(2, (char*)"CUDA-normal");
    timer.setTimerName(3, (char*)"CUDA-shared memory");



    cout << "�̹��� ���ϸ��� �Է����ּ��� : ";
    cin >> fileName;
    original_image = imread(fileName, IMREAD_COLOR);
    if (original_image.data == NULL) {
        cout << "�̹����� ã�� ���߽��ϴ�.";
        getchar();
        getchar();
        return -1;
    }

    unsigned char* input = (unsigned char*)(original_image.data);
    unsigned char* dev_input, * dev_output;
    unsigned char* output = (unsigned char*)malloc(original_image.cols * original_image.rows * 3 * sizeof(char));
    unsigned char* output_sm = (unsigned char*)malloc(original_image.cols * original_image.rows * 3 * sizeof(char));




    hipMalloc(&original_image_cuda, sizeof(Mat));

    result_serial = Mat::zeros(original_image.size(), original_image.type());

    result_openmp = Mat::zeros(original_image.size(), original_image.type());

    timer.onTimer(0);
    mosaic_serial(&original_image, &result_serial, mosaic);
    timer.offTimer(0);

    timer.onTimer(1);
    mosaic_openmp(&original_image, &result_openmp, mosaic);
    timer.offTimer(1);




    int imgh = original_image.size().height;
    int imgw = original_image.size().width;

    int size = original_image.size().height * original_image.size().width;

    dim3 dimBlock(32, 16);
    dim3 dimGrid(ceil((float)imgh / dimBlock.x), ceil((float)imgw / dimBlock.y), 1);





    timer.onTimer(2);
    hipMalloc((void**)&dev_input, original_image.cols * original_image.rows * 3 * sizeof(char));
    hipMalloc((void**)&dev_output, original_image.cols * original_image.rows * 3 * sizeof(char));
    hipMemcpy(dev_input, input, original_image.cols * original_image.rows * 3 * sizeof(char), hipMemcpyHostToDevice);
    
    mosaic_cuda << <dimGrid, dimBlock >> > (dev_input, dev_output, mosaic, imgh, imgw);
    hipDeviceSynchronize();
    
    hipMemcpy(output, dev_output, original_image.cols * original_image.rows * 3 * sizeof(char), hipMemcpyDeviceToHost);
    timer.offTimer(2);




    Mat file3 = Mat(original_image.rows, original_image.cols, original_image.type(), output);

    //------------------------------


    timer.onTimer(3);
    hipMalloc((void**)&dev_input, original_image.cols * original_image.rows * 3 * sizeof(char));
    hipMalloc((void**)&dev_output, original_image.cols * original_image.rows * 3 * sizeof(char));
    hipMemcpy(dev_input, input, original_image.cols * original_image.rows * 3 * sizeof(char), hipMemcpyHostToDevice);
   
    mosaic_cuda_sm << <dimGrid, dimBlock >> > (dev_input, dev_output, mosaic, imgh, imgw);
    hipDeviceSynchronize();
    
    hipMemcpy(output_sm, dev_output, original_image.cols * original_image.rows * 3 * sizeof(char), hipMemcpyDeviceToHost);
    timer.offTimer(3);


    Mat file4 = Mat(original_image.rows, original_image.cols, original_image.type(), output_sm);




    /* show image */
    namedWindow("serial", WINDOW_NORMAL);
    imshow("serial", result_serial);

    namedWindow("openmp", WINDOW_NORMAL);
    imshow("openmp", result_openmp);

    namedWindow("CUDA-normal", WINDOW_NORMAL);
    imshow("CUDA-normal", file3);

    namedWindow("CUDA-shared memory", WINDOW_NORMAL);
    imshow("CUDA-shared memory", file4);


    timer.printTimer();
    waitKey();
    destroyAllWindows();





}